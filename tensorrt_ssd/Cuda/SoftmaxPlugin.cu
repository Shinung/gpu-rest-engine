#include "plugin/SoftmaxPlugin.h"
#include "cudaUtility.h"

using namespace trt::cudnn;

int SoftmaxPlugin::enqueue(int batchSize, const void* const* inputs, void** outputs, void* workspace, hipStream_t stream)
{
    //LOG(INFO) << "bottom_desc_ set";
    CUDNN_CHECK(hipdnnSetTensor4dDescriptorEx(bottom_desc_, dataType<float>::type, 
                                            mBottomH, mBottomC, 1, 1,
                                            mBottomC, 1, 1, 1));
    //LOG(INFO) << "top_desc_ set";
    CUDNN_CHECK(hipdnnSetTensor4dDescriptorEx(top_desc_, dataType<float>::type, 
                                            mBottomH, mBottomC, 1, 1,
                                            mBottomC, 1, 1, 1));
    
    CUDNN_CHECK(hipdnnSoftmaxForward(handle_, HIPDNN_SOFTMAX_ACCURATE,
        HIPDNN_SOFTMAX_MODE_CHANNEL,
        dataType<float>::one,
        bottom_desc_, *inputs,
        dataType<float>::zero,
        top_desc_, *outputs));
    
    return 0;
}